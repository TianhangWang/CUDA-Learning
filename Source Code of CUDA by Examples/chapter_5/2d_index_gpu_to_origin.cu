#include "hip/hip_runtime.h"
#include <iostream>
#include "..depend/cpu_anim.h"
#define DIM 512

struct DataBlock{
    unsigned char *dev_bitmap;
    CPUAnimBitmap *bitmap;
};

void cleanup (DataBlock *d){
    hipFree(d->dev_bitmap);
}

__global__ void kernel(unsigned char *ptr, int ticks){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    // 这两步将 block + thread 组成的索引映射到对应二维网格上;
    int offset = x + y * blockDim.x * gridDim.x;
    // 这部将二维网格映射到 1-D array 上;
    
    // now calculate the value at that position
    float fx = x - DIM/2;
    float fy = y - DIM/2;
    float d = sqrtf( fx * fx + fy * fy );

    unsigned char grey = (unsigned char)(128.0f + 127.0f *
                                         cos(d/10.0f - ticks/7.0f) /
                                         (d/10.0f + 1.0f));    
    ptr[offset*4 + 0] = grey;
    ptr[offset*4 + 1] = grey;
    ptr[offset*4 + 2] = grey;
    ptr[offset*4 + 3] = 255; 
}

void generate_frame(DataBlock *d, int ticks){
    dim3 blocks(DIM/16, DIM/16);
    dim3 threads(16,16);

    kernel<<<blocks,threads>>>(d->dev_bitmap, ticks);

    hipMemcpy(d->bitmap->get_ptr(),
               d->dev_bitmap,
               d->bitmap->image_size(),
               hipMemcpyDeviceToHost);
}

int main(){
    DataBlock data;
    CPUAnimBitmap bitmap(DIM, DIM, &data);
    data.bitmap = &bitmap;

    hipMalloc((void **)&data.dev_bitmap, bitmap.image_size());

    bitmap.anim_and_exit((void (*)(void*, int))generate_frame,
                         (void (*)(void*))cleanup);
}